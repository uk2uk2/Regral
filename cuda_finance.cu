#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <string>
#include <hip/hip_runtime.h>

// ------------------------------------------------------
// CUDA Kernel Example: Compute Daily Returns
// For an array of prices: returns[i] = (price[i+1] - price[i]) / price[i]
// ------------------------------------------------------
__global__
void dailyReturnsKernel(const float* prices, float* returns, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n - 1) {
        float pToday = prices[idx];
        float pTomorrow = prices[idx + 1];
        // Basic daily return formula (watch out for dividing by zero in real usage).
        returns[idx] = (pTomorrow - pToday) / pToday;
    }
}

int main(int argc, char* argv[])
{
    if (argc < 2) {
        std::cerr << "Usage: ./cuda_finance <csv_file>" << std::endl;
        return 1;
    }
    std::string filename = argv[1];

    // ------------------------------------------------------
    // Step 1: Read CSV into Host Vector
    // ------------------------------------------------------
    std::vector<float> prices;
    {
        std::ifstream file(filename);
        if (!file.is_open()) {
            std::cerr << "Could not open file: " << filename << std::endl;
            return 1;
        }

        // Example: CSV with a single column of prices or 
        // possibly more columns where the 2nd col is price. 
        // For demonstration, assume 1st row is a header we skip.
        std::string line;
        bool skipHeader = true;
        while (std::getline(file, line)) {
            if (skipHeader) {
                skipHeader = false;
                continue;
            }
            std::stringstream ss(line);
            std::string valStr;
            // Here, assume each line has at least one numeric field we want
            if (std::getline(ss, valStr, ',')) {
                try {
                    float price = std::stof(valStr);
                    prices.push_back(price);
                } catch (...) {
                    // If parsing fails, skip or handle error
                }
            }
        }
        file.close();
    }
    int n = prices.size();
    if (n < 2) {
        std::cerr << "Not enough price data to compute returns." << std::endl;
        return 1;
    }
    std::cout << "Loaded " << n << " price entries from CSV." << std::endl;

    // ------------------------------------------------------
    // Step 2: Allocate Device Memory for Prices and Returns
    // ------------------------------------------------------
    float *d_prices = nullptr;
    float *d_returns = nullptr;
    hipMalloc((void**)&d_prices, n * sizeof(float));
    hipMalloc((void**)&d_returns, n * sizeof(float));  // same length, though last is not used

    // ------------------------------------------------------
    // Step 3: Copy Data from Host to Device
    // ------------------------------------------------------
    hipMemcpy(d_prices, prices.data(), n * sizeof(float), hipMemcpyHostToDevice);

    // ------------------------------------------------------
    // Step 4: Launch Kernel to Process Data in Parallel
    // ------------------------------------------------------
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;
    dailyReturnsKernel<<<gridSize, blockSize>>>(d_prices, d_returns, n);
    hipDeviceSynchronize();

    // ------------------------------------------------------
    // Step 5: Copy Results Back to Host
    // ------------------------------------------------------
    std::vector<float> returns(n);
    hipMemcpy(returns.data(), d_returns, n * sizeof(float), hipMemcpyDeviceToHost);

    // (Optional) Print some sample outputs
    std::cout << "Sample computed returns (first 5):" << std::endl;
    for (int i = 0; i < 5 && i < n - 1; i++) {
        std::cout << "Day " << i << " -> " << returns[i] << std::endl;
    }

    // ------------------------------------------------------
    // Cleanup
    // ------------------------------------------------------
    hipFree(d_prices);
    hipFree(d_returns);

    return 0;
}
